#include "hip/hip_runtime.h"
extern "C" {
	#include "gpu.h"
}

static hipCtx_t cuContext;

static void checkCuError(hipError_t err, const char *msg)
{
	if (err != hipSuccess) {
		printf("Cuda Error: %s\n", msg);
		exit(EXIT_FAILURE);
	}
}

static void checkCudaError(hipError_t err, const char *msg)
{
	if (err != hipSuccess) {
		printf("%s: %s\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

static int get_gpu_device_id(const char *bdf)
{
	return 0;
}

void *init_gpu(size_t size, const char *bdf)
{
    const size_t gpu_page_size = 64 * 1024;
    hipDevice_t cu_dev;
    hipDeviceptr_t d_A;

    size_t aligned_size = (size + gpu_page_size - 1) & ~(gpu_page_size - 1);

    int dev_id = get_gpu_device_id(bdf);
    if (dev_id < 0) {
        printf("Wrong device index (%d) obtained from bdf \"%s\"\n",
                dev_id, bdf);
        return NULL;
    }

    checkCuError(hipInit(0), "hipInit failed");
    checkCuError(hipDeviceGet(&cu_dev, dev_id), "hipDeviceGet failed");
    checkCuError(hipCtxCreate(&cuContext, hipDeviceMapHost, cu_dev), 
			"hipCtxCreate failed");
    checkCuError(hipMalloc(&d_A, aligned_size), "hipMalloc failed");

    return ((void *) d_A);
}

int free_gpu(void *gpu_buf)
{
    hipDeviceptr_t d_A = (hipDeviceptr_t) gpu_buf;

    hipFree(d_A);
    d_A = 0;

    checkCuError(hipCtxDestroy(cuContext), "hipCtxDestroy failed");

    return 0;
}

void gpuMemAlloc(void **buf, int size)
{
	checkCudaError(hipMalloc(buf, size), "hipMalloc failed");
}

void gpuMemFree(float *buf)
{
	hipFree(buf);
}

void gpuPinnedMemAlloc(void **buf, int size)
{
	checkCudaError(hipHostAlloc(buf, size, hipHostMallocMapped),
			"hipHostAlloc failed");
}

void gpuPinnedMemMap(void **gpu, void **host)
{
	checkCudaError(hipHostGetDevicePointer(gpu, host, 0),
			"hipHostGetDevicePointer failed");
}

void gpuPinnedMemFree(float *buf)
{
	hipHostFree(buf);
}

/* Simple GPU kernel */
static __global__ void squareKernel(float *data, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		data[idx] *= data[idx];
	}
}

int inferCudaMemcpy(float *host, float *gpu, int size)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

	/* Host -> GPU memory copy */
	checkCudaError(hipMemcpy(gpu, host, size, hipMemcpyHostToDevice),
			"hipMemcpy failed");

	/* Inference */
	squareKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu, size);
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("Cuda Error: %s\n", hipGetErrorString(err));
	}

	/* GPU -> Host memory copy */
	checkCudaError(hipMemcpy(host, gpu, size, hipMemcpyDeviceToHost),
			"hipMemcpy failed");

	return 0;
}

int inferZeroCopy(float *gpu, int size)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

	/* Inference */
	squareKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu, size);
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("Cuda Error: %s\n", hipGetErrorString(err));
	}

	return 0;
}

// Initialize the Python interpreter and load the module
InferenceContext *initialize_inference(const char *module_name) {
    Py_Initialize();

    PyObject *sysPath = PySys_GetObject("path");
    PyObject *currentDir = PyUnicode_FromString(".");
    PyList_Append(sysPath, currentDir);
    Py_DECREF(currentDir);

    PyObject *pName = PyUnicode_FromString(module_name);
    PyObject *pModule = PyImport_Import(pName);
    Py_DECREF(pName);

    if (!pModule) {
        PyErr_Print();
        Py_Finalize();
        return NULL;
    }

    InferenceContext *ctx = (InferenceContext *) 
		malloc(sizeof(InferenceContext));
    ctx->module = pModule;
    ctx->model = NULL;

    return ctx;
}

// Load the model
int load_model(InferenceContext *ctx) {
    PyObject *pFuncLoadModel = 
		PyObject_GetAttrString(ctx->module, "load_model");
    if (!pFuncLoadModel || !PyCallable_Check(pFuncLoadModel)) {
        PyErr_Print();
        return -1;
    }

    ctx->model = PyObject_CallObject(pFuncLoadModel, NULL);
    Py_DECREF(pFuncLoadModel);

    if (!ctx->model) {
        PyErr_Print();
        return -1;
    }

    return 0;
}

// Allocate GPU memory for input and output tensor
InputTensor *allocate_gpu_memory(size_t size) {
    InputTensor *input = (InputTensor *) malloc(sizeof(InputTensor));
    if (!input) {
        fprintf(stderr, "Failed to allocate InputTensor structure\n");
        return NULL;
    }

    if (hipMalloc(&input->gpu_data, size) != hipSuccess) {
        fprintf(stderr, "Failed to allocate GPU memory\n");
        free(input);
        return NULL;
    }
    input->size = size;
    input->tensor = NULL;

    return input;
}

// Preprocess data on CPU
PyObject *preprocess_on_cpu(InferenceContext *ctx, 
		const void *cpu_data, size_t size) {
    PyObject *pFuncPreprocess = 
		PyObject_GetAttrString(ctx->module, "preprocess");
    if (!pFuncPreprocess || !PyCallable_Check(pFuncPreprocess)) {
        PyErr_Print();
        return NULL;
    }

    PyObject *pCpuData = 
		PyMemoryView_FromMemory((char *) cpu_data, size, PyBUF_READ);
	size_t num_floats = size / sizeof(float);
	PyObject *pSize = PyLong_FromSize_t(num_floats);
    PyObject *pArgsPreprocess = PyTuple_Pack(2, pCpuData, pSize);

    PyObject *pProcessedTensor = 
		PyObject_CallObject(pFuncPreprocess, pArgsPreprocess);
    Py_DECREF(pCpuData);
    Py_DECREF(pArgsPreprocess);
    Py_DECREF(pFuncPreprocess);

    if (!pProcessedTensor) {
        PyErr_Print();
        return NULL;
    }

    return pProcessedTensor;
}

// Copy CPU processed data to GPU
int copy_to_gpu(InputTensor *input, PyObject *pProcessedTensor) {
    void *cpu_memory = PyMemoryView_GET_BUFFER(pProcessedTensor);
    if (hipMemcpy(input->gpu_data, cpu_memory, input->size, 
				hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Failed to copy preprocessed data to GPU memory\n");
        Py_DECREF(pProcessedTensor);
        return -1;
    }

    input->tensor = pProcessedTensor;

    return 0;
}

// Perform inference and keep the result in GPU memory
InputTensor *run_inference(InferenceContext *ctx, InputTensor *input) {
    PyObject *pFuncInfer = PyObject_GetAttrString(ctx->module, "infer");
    if (!pFuncInfer || !PyCallable_Check(pFuncInfer)) {
        PyErr_Print();
        return NULL;
    }

    PyObject *pArgsInfer = PyTuple_Pack(2, ctx->model, input->tensor);
    PyObject *pOutputGpu = PyObject_CallObject(pFuncInfer, pArgsInfer);
    Py_DECREF(pArgsInfer);
    Py_DECREF(pFuncInfer);

    if (!pOutputGpu) {
        PyErr_Print();
        return NULL;
    }

    InputTensor *output = (InputTensor *) malloc(sizeof(InputTensor));
    output->tensor = pOutputGpu;
    output->gpu_data = NULL;
    output->size = 0;

    return output;
}

// Postprocess the inference result on CPU
PyObject *postprocess_on_cpu(InferenceContext *ctx, PyObject *output_tensor) {
    PyObject *pFuncPostprocess = 
		PyObject_GetAttrString(ctx->module, "postprocess");
    if (!pFuncPostprocess || !PyCallable_Check(pFuncPostprocess)) {
        PyErr_Print();
        return NULL;
    }

    PyObject *pArgsPostprocess = PyTuple_Pack(1, output_tensor);
    PyObject *pProcessedResult = PyObject_CallObject(pFuncPostprocess, 
			pArgsPostprocess);
    Py_DECREF(pArgsPostprocess);
    Py_DECREF(pFuncPostprocess);

    if (!pProcessedResult) {
        PyErr_Print();
        return NULL;
    }

    return pProcessedResult;
}

// Free resources
void free_tensor(InputTensor *tensor) {
    if (tensor) {
        if (tensor->gpu_data) hipFree(tensor->gpu_data);
        if (tensor->tensor) Py_DECREF(tensor->tensor);
        free(tensor);
    }
}

void free_inference_context(InferenceContext *ctx) {
    if (ctx) {
        if (ctx->model) Py_DECREF(ctx->model);
        Py_DECREF(ctx->module);
        free(ctx);
    }

    Py_Finalize();
}

