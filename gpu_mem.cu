#include "hip/hip_runtime.h"
#include "gpu_mem.h"

static hipCtx_t cuContext;

static void checkCuError(hipError_t err, const char *msg)
{
	if (err != hipSuccess) {
		printf("Cuda Error: %s\n", msg);
		exit(EXIT_FAILURE);
	}
}

static void checkCudaError(hipError_t err, const char *msg)
{
	if (err != hipSuccess) {
		printf("%s: %s\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

static int get_gpu_device_id(const char *bdf)
{
	return 0;
}

void *init_gpu(size_t size, const char *bdf)
{
    const size_t gpu_page_size = 64 * 1024;
    hipDevice_t cu_dev;
    hipDeviceptr_t d_A;

    size_t aligned_size = (size + gpu_page_size - 1) & ~(gpu_page_size - 1);

    int dev_id = get_gpu_device_id(bdf);
    if (dev_id < 0) {
        printf("Wrong device index (%d) obtained from bdf \"%s\"\n",
                dev_id, bdf);
        return NULL;
    }

    checkCuError(hipInit(0), "hipInit failed");
    checkCuError(hipDeviceGet(&cu_dev, dev_id), "hipDeviceGet failed");
    checkCuError(hipCtxCreate(&cuContext, hipDeviceMapHost, cu_dev), 
			"hipCtxCreate failed");
    checkCuError(hipMalloc(&d_A, aligned_size), "hipMalloc failed");

    return ((void *) d_A);
}

int free_gpu(void *gpu_buf)
{
    hipDeviceptr_t d_A = (hipDeviceptr_t) gpu_buf;

    hipFree(d_A);
    d_A = 0;

    checkCuError(hipCtxDestroy(cuContext), "hipCtxDestroy failed");

    return 0;
}

void gpuMemAlloc(void **buf, int size)
{
	checkCudaError(hipMalloc(buf, size), "hipMalloc failed");
}

void gpuMemFree(float *buf)
{
	hipFree(buf);
}

void gpuPinnedMemAlloc(void **buf, int size)
{
	checkCudaError(hipHostAlloc(buf, size, hipHostMallocMapped),
			"hipHostAlloc failed");
}

void gpuPinnedMemMap(void **gpu, void **host)
{
	checkCudaError(hipHostGetDevicePointer(gpu, host, 0),
			"hipHostGetDevicePointer failed");
}

void gpuPinnedMemFree(float *buf)
{
	hipHostFree(buf);
}

/* Simple GPU kernel */
static __global__ void squareKernel(float *data, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		data[idx] *= data[idx];
	}
}

int inferCudaMemcpy(float *host, float *gpu, int size)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

	/* Host -> GPU memory copy */
	checkCudaError(hipMemcpy(gpu, host, size, hipMemcpyHostToDevice),
			"hipMemcpy failed");

	/* Inference */
	squareKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu, size);
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("Cuda Error: %s\n", hipGetErrorString(err));
	}

	/* GPU -> Host memory copy */
	checkCudaError(hipMemcpy(host, gpu, size, hipMemcpyDeviceToHost),
			"hipMemcpy failed");

	return 0;
}

int inferZeroCopy(float *gpu, int size)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

	/* Inference */
	squareKernel<<<blocksPerGrid, threadsPerBlock>>>(gpu, size);
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("Cuda Error: %s\n", hipGetErrorString(err));
	}

	return 0;
}

